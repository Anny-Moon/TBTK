#include "hip/hip_runtime.h"
/** @file ChebyshevSolver.cu
 *
 *  @author Kristofer Björnson
 */

#include "../include/ChebyshevSolver.h"
#include <math.h>
#include "../include/HALinkedList.h"
#include <hip/hip_complex.h>

using namespace std;

namespace TBTK{

__global__
void multiplyMatrixAndVector(hipDoubleComplex *jIn,
				hipDoubleComplex *jResult,
				hipDoubleComplex *hoppingAmplitudes,
				int *fromIndices,
				int maxHoppingAmplitudes,
				int basisSize,
				hipDoubleComplex *coefficients,
				int currentCoefficient,
				int *coefficientMap,
				int numCoefficients){
	int to = blockIdx.x*blockDim.x + threadIdx.x;
	if(to < basisSize)
		for(int n = 0; n < maxHoppingAmplitudes; n++)
			jResult[to] = hipCadd(jResult[to], hipCmul(hoppingAmplitudes[maxHoppingAmplitudes*to + n], jIn[fromIndices[maxHoppingAmplitudes*to + n]]));

/*	if(to == coefficientIndex)
		coefficients[currentCoefficient] = jResult[to];*/
	if(to < basisSize && coefficientMap[to] != -1)
		coefficients[coefficientMap[to]*numCoefficients + currentCoefficient] = jResult[to];
}

__global__
void subtractVector(hipDoubleComplex *jIn2, hipDoubleComplex *jResult, int basisSize){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < basisSize)
		jResult[idx] = make_hipDoubleComplex(-hipCreal(jIn2[idx]), -hipCimag(jIn2[idx]));
}

void debugCUDA(complex<double> *jIn1_device, complex<double> *jIn2_device, complex<double> *jResult_device, int basisSize){
	complex<double> *jIn1 = new complex<double>[basisSize];
	complex<double> *jIn2 = new complex<double>[basisSize];
	complex<double> *jResult = new complex<double>[basisSize];
	hipMemcpy(jIn1, jIn1_device, basisSize*sizeof(complex<double>), hipMemcpyDeviceToHost);
	hipMemcpy(jIn2, jIn2_device, basisSize*sizeof(complex<double>), hipMemcpyDeviceToHost);
	hipMemcpy(jResult, jResult_device, basisSize*sizeof(complex<double>), hipMemcpyDeviceToHost);
	for(int n = 0; n < basisSize; n++)
		cout << n << "\t" << jIn1[n] << "\t" << jIn2[n] << "\t" << jResult[n] << "\n";

	delete [] jIn1;
	delete [] jIn2;
	delete [] jResult;
}

void debugNormal(complex<double> *jIn1, complex<double> *jIn2, complex<double> *jResult, int basisSize){
	for(int n = 0; n < basisSize; n++)
		cout << n << "\t" << jIn1[n] << "\t" << jIn2[n] << "\t" << jResult[n] << "\n";
}

void ChebyshevSolver::calculateCoefficientsGPU(Index to, Index from, complex<double> *coefficients, int numCoefficients, double broadening){
	vector<Index> toVector;
	toVector.push_back(to);
	calculateCoefficientsGPU(toVector, from, coefficients, numCoefficients, broadening);
}

void ChebyshevSolver::calculateCoefficientsGPU(vector<Index> &to, Index from, complex<double> *coefficients, int numCoefficients, double broadening){
	AmplitudeSet *amplitudeSet = &model->amplitudeSet;

	int fromBasisIndex = amplitudeSet->getBasisIndex(from);
	int *coefficientMap = new int[amplitudeSet->getBasisSize()];
	for(int n = 0; n < amplitudeSet->getBasisSize(); n++)
		coefficientMap[n] = -1;
	for(int n = 0; n < to.size(); n++)
		coefficientMap[amplitudeSet->getBasisIndex(to.at(n))] = n;

	if(isTalkative){
		cout << "ChebyshevSolver::calculateCoefficientsGPU\n";
		cout << "\tFrom Index: " << fromBasisIndex << "\n";
		cout << "\tBasis size: " << amplitudeSet->getBasisSize() << "\n";
	}

	complex<double> *jIn1 = new complex<double>[amplitudeSet->getBasisSize()];
	complex<double> *jIn2 = new complex<double>[amplitudeSet->getBasisSize()];
	complex<double> *jResult = new complex<double>[amplitudeSet->getBasisSize()];
	complex<double> *jTemp = NULL;
	for(int n = 0; n < amplitudeSet->getBasisSize(); n++){
		jIn1[n] = 0.;
		jIn2[n] = 0.;
		jResult[n] = 0.;
	}

	//Set up initial state (|j0>)
	jIn1[fromBasisIndex] = 1.;

	for(int n = 0; n < amplitudeSet->getBasisSize(); n++)
		if(coefficientMap[n] != -1)
			coefficients[coefficientMap[n]*numCoefficients] = jIn1[n];

	//Generate a fixed hopping amplitude and index list, for speed.
	AmplitudeSet::iterator it = amplitudeSet->getIterator();
	HoppingAmplitude *ha;
	int *numHoppingAmplitudes = new int[amplitudeSet->getBasisSize()];
	for(int n = 0; n < amplitudeSet->getBasisSize(); n++)
		numHoppingAmplitudes[n] = 0;
	while((ha = it.getHA())){
		numHoppingAmplitudes[amplitudeSet->getBasisIndex(ha->toIndex)]++;
		it.searchNextHA();
	}
	int maxHoppingAmplitudes = 0;
	for(int n = 0; n < amplitudeSet->getBasisSize(); n++)
		if(numHoppingAmplitudes[n] > maxHoppingAmplitudes)
			maxHoppingAmplitudes = numHoppingAmplitudes[n];

	delete [] numHoppingAmplitudes;

	int *currentHoppingAmplitudes = new int[amplitudeSet->getBasisSize()];
	for(int n = 0; n < amplitudeSet->getBasisSize(); n++)
		currentHoppingAmplitudes[n] = 0;

	complex<double> *hoppingAmplitudes = new complex<double>[maxHoppingAmplitudes*amplitudeSet->getBasisSize()];
	int *fromIndices = new int[maxHoppingAmplitudes*amplitudeSet->getBasisSize()];
	for(int n = 0; n < maxHoppingAmplitudes*amplitudeSet->getBasisSize(); n++){
		hoppingAmplitudes[n] = 0.;
		fromIndices[n] = 0;
	}

	it.reset();
	while((ha = it.getHA())){
		int to = amplitudeSet->getBasisIndex(ha->toIndex);
		int from = amplitudeSet->getBasisIndex(ha->fromIndex);

		hoppingAmplitudes[maxHoppingAmplitudes*to + currentHoppingAmplitudes[to]] = ha->getAmplitude()/scaleFactor;
		fromIndices[maxHoppingAmplitudes*to + currentHoppingAmplitudes[to]] = from;

		currentHoppingAmplitudes[to]++;

		it.searchNextHA();
	}

	delete [] currentHoppingAmplitudes;

	//Initialize GPU
	complex<double> *jIn1_device;
	complex<double> *jIn2_device;
	complex<double> *jResult_device;
	complex<double> *hoppingAmplitudes_device;
	int *fromIndices_device;
	complex<double> *coefficients_device;
	int *coefficientMap_device;

	int totalMemoryRequirement = amplitudeSet->getBasisSize()*sizeof(complex<double>);
	totalMemoryRequirement += amplitudeSet->getBasisSize()*sizeof(complex<double>);
	totalMemoryRequirement += amplitudeSet->getBasisSize()*sizeof(complex<double>);
	totalMemoryRequirement += maxHoppingAmplitudes*amplitudeSet->getBasisSize()*sizeof(complex<double>);
	totalMemoryRequirement += maxHoppingAmplitudes*amplitudeSet->getBasisSize()*sizeof(int);
	totalMemoryRequirement += to.size()*numCoefficients*sizeof(complex<double>);
	totalMemoryRequirement += amplitudeSet->getBasisSize()*sizeof(int);
	if(isTalkative){
		cout << "\tCUDA memory requirement: ";
		if(totalMemoryRequirement < 1024)
			cout << totalMemoryRequirement/1024 << "B\n";
		else if(totalMemoryRequirement < 1024*1024)
			cout << totalMemoryRequirement/1024 << "KB\n";
		else
			cout << totalMemoryRequirement/1024/1024 << "MB\n";
	}

	if(hipMalloc((void**)&jIn1_device, amplitudeSet->getBasisSize()*sizeof(complex<double>)) != hipSuccess)
		{	cout << "\tMalloc error\n";	exit(1);	}
	if(hipMalloc((void**)&jIn2_device, amplitudeSet->getBasisSize()*sizeof(complex<double>)) != hipSuccess)
		{	cout << "\tMalloc error\n";	exit(1);	}
	if(hipMalloc((void**)&jResult_device, amplitudeSet->getBasisSize()*sizeof(complex<double>)) != hipSuccess)
		{	cout << "\tMalloc error\n";	exit(1);	}
	if(hipMalloc((void**)&hoppingAmplitudes_device, maxHoppingAmplitudes*amplitudeSet->getBasisSize()*sizeof(complex<double>)) != hipSuccess)
		{	cout << "\tMalloc error\n";	exit(1);	}
	if(hipMalloc((void**)&fromIndices_device, maxHoppingAmplitudes*amplitudeSet->getBasisSize()*sizeof(int)) != hipSuccess)
		{	cout << "\tMalloc error\n";	exit(1);	}
	if(hipMalloc((void**)&coefficients_device, to.size()*numCoefficients*sizeof(complex<double>)) != hipSuccess)
		{	cout << "\tMalloc error\n";	exit(1);	}
	if(hipMalloc((void**)&coefficientMap_device, amplitudeSet->getBasisSize()*sizeof(int)) != hipSuccess)
		{	cout << "\tMalloc error\n";	exit(1);	}

	if(hipMemcpy(jIn1_device, jIn1, amplitudeSet->getBasisSize()*sizeof(complex<double>), hipMemcpyHostToDevice) != hipSuccess)
		{	cout << "\tMemcpy error\n";	exit(1);	}
	if(hipMemcpy(jIn2_device, jIn2, amplitudeSet->getBasisSize()*sizeof(complex<double>), hipMemcpyHostToDevice) != hipSuccess)
		{	cout << "\tMemcpy error\n";	exit(1);	}
	if(hipMemcpy(jResult_device, jResult, amplitudeSet->getBasisSize()*sizeof(complex<double>), hipMemcpyHostToDevice) != hipSuccess)
		{	cout << "\tMemcpy error\n";	exit(1);	}
	if(hipMemcpy(hoppingAmplitudes_device, hoppingAmplitudes, maxHoppingAmplitudes*amplitudeSet->getBasisSize()*sizeof(complex<double>), hipMemcpyHostToDevice) != hipSuccess)
		{	cout << "\tMemcpy error\n";	exit(1);	}
	if(hipMemcpy(fromIndices_device, fromIndices, maxHoppingAmplitudes*amplitudeSet->getBasisSize()*sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
		{	cout << "\tMemcpy error\n";	exit(1);	}
	if(hipMemcpy(coefficients_device, coefficients, to.size()*numCoefficients*sizeof(complex<double>), hipMemcpyHostToDevice) != hipSuccess)
		{	cout << "\tMemcpy error\n";	exit(1);	}
	if(hipMemcpy(coefficientMap_device, coefficientMap, amplitudeSet->getBasisSize()*sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
		{	cout << "\tMemcpy error\n";	exit(1);	}

	//Calculate |j1>
	int block_size = 1024;
	int num_blocks = amplitudeSet->getBasisSize()/block_size + (amplitudeSet->getBasisSize()%block_size == 0 ? 0:1);
	if(isTalkative){
		cout << "\tCUDA Block size: " << block_size << "\n";
		cout << "\tCUDA Num blocks: " << num_blocks << "\n";
	}
	multiplyMatrixAndVector <<< num_blocks, block_size>>> ((hipDoubleComplex*)jIn1_device,
								(hipDoubleComplex*)jResult_device,
								(hipDoubleComplex*)hoppingAmplitudes_device,
								fromIndices_device,
								maxHoppingAmplitudes,
								amplitudeSet->getBasisSize(),
								(hipDoubleComplex*)coefficients_device,
								1,
								coefficientMap_device,
								numCoefficients);
	hipError_t code = hipGetLastError();
	if(code != hipSuccess){
		cout << "\tMatrix vector multiplication error 1\n";
		cout << "\t" << hipGetErrorString(code) << "\n";
		cout << "\tCUDA Block size: " << block_size << "\n";
		cout << "\tCUDA Num blocks: " << num_blocks << "\n";
		exit(1);
	}

	jTemp = jIn2_device;
	jIn2_device = jIn1_device;
	jIn1_device = jResult_device;
	jResult_device = jTemp;

	//Multiply hopping amplitudes by factor two, to speed up calculation of 2H|j(n-1)> - |j(n-2)>.
	for(int n = 0; n < maxHoppingAmplitudes*amplitudeSet->getBasisSize(); n++)
		hoppingAmplitudes[n] *= 2.;
	hipMemcpy(hoppingAmplitudes_device, hoppingAmplitudes, maxHoppingAmplitudes*amplitudeSet->getBasisSize()*sizeof(complex<double>), hipMemcpyHostToDevice);

	if(isTalkative)
		cout << "\tProgress (100 coefficients per dot): ";

	//Iteratively calculate |jn> and corresponding Chebyshev coefficients.
	for(int n = 2; n < numCoefficients; n++){
		subtractVector <<< num_blocks, block_size >>> ((hipDoubleComplex*)jIn2_device,
								(hipDoubleComplex*)jResult_device,
								amplitudeSet->getBasisSize());
		if(hipGetLastError() != hipSuccess){	cout << "Subtraction error\n";	exit(1);	}

		multiplyMatrixAndVector <<< num_blocks, block_size>>> ((hipDoubleComplex*)jIn1_device,
									(hipDoubleComplex*)jResult_device,
									(hipDoubleComplex*)hoppingAmplitudes_device,
									fromIndices_device,
									maxHoppingAmplitudes,
									amplitudeSet->getBasisSize(),
									(hipDoubleComplex*)coefficients_device,
									n,
									coefficientMap_device,
									numCoefficients);
		if(hipGetLastError() != hipSuccess){	cout << "Matrix vector multiplication error 2\n";	exit(1);	}

		jTemp = jIn2_device;
		jIn2_device = jIn1_device;
		jIn1_device = jResult_device;
		jResult_device = jTemp;

		if(isTalkative){
			if(n%100 == 0)
				cout << "." << flush;
			if(n%1000 == 0)
				cout << " " << flush;
		}
	}
	if(isTalkative)
		cout << "\n";

	if(hipMemcpy(coefficients, coefficients_device, to.size()*numCoefficients*sizeof(complex<double>), hipMemcpyDeviceToHost) != hipSuccess){
		cout << "\tMemcpy error\n";
		exit(1);
	}

	delete [] jIn1;
	delete [] jIn2;
	delete [] jResult;
	delete [] hoppingAmplitudes;
	delete [] fromIndices;
	delete [] coefficientMap;

	hipFree(jIn1_device);
	hipFree(jIn2_device);
	hipFree(jResult_device);
	hipFree(hoppingAmplitudes_device);
	hipFree(fromIndices_device);
	hipFree(coefficients_device);
	hipFree(coefficientMap_device);

	//Lorentzian convolution
	double lambda = broadening*numCoefficients;
	for(int n = 0; n < numCoefficients; n++)
		for(int c = 0; c < to.size(); c++)
			coefficients[n + c*numCoefficients] = coefficients[n + c*numCoefficients]*sinh(lambda*(1 - n/(double)numCoefficients))/sinh(lambda);
}

__global__
void calculateGreensFunction(hipDoubleComplex *greensFunction,
				hipDoubleComplex *coefficients,
				hipDoubleComplex *lookupTable,
				int numCoefficients,
				int energyResolution){
	int e = blockIdx.x*blockDim.x + threadIdx.x;
	if(e < energyResolution)
		for(int n = 0; n < numCoefficients; n++)
			greensFunction[e] = hipCadd(greensFunction[e], hipCmul(lookupTable[n*energyResolution + e], coefficients[n]));
//			greensFunction[e] += lookupTable[n*energyResolution + e]*coefficients[n];
}

void ChebyshevSolver::loadLookupTableGPU(){
	if(isTalkative)
		cout << "CheyshevSolver::loadLookupTableGPU\n";

	if(generatingFunctionLookupTable == NULL){
		cout << "\tError: Lookup table has not been generated.\n";
		exit(1);
	}
	if(generatingFunctionLookupTable_device != NULL){
		cout << "\tError: Lookup table already loaded.\n";
		exit(1);
	}

	complex<double> *generatingFunctionLookupTable_host = new complex<double>[lookupTableNumCoefficients*lookupTableResolution];
	for(int n = 0; n < lookupTableNumCoefficients; n++)
		for(int e = 0; e < lookupTableResolution; e++)
			generatingFunctionLookupTable_host[n*lookupTableResolution + e] = generatingFunctionLookupTable[n][e];

	int memoryRequirement = lookupTableNumCoefficients*lookupTableResolution*sizeof(complex<double>);
	if(isTalkative){
		cout << "\tCUDA memory requirement: ";
		if(memoryRequirement < 1024)
			cout << memoryRequirement << "B";
		else if(memoryRequirement < 1024*1024)
			cout << memoryRequirement/1024 << "KB";
		else
			cout << memoryRequirement/1024/1024 << "MB";
	}

	if(hipMalloc((void**)&generatingFunctionLookupTable_device, lookupTableNumCoefficients*lookupTableResolution*sizeof(complex<double>))  != hipSuccess)
		{	cout << "\tMalloc error\n";	exit(1);	}

	if(hipMemcpy(generatingFunctionLookupTable_device, generatingFunctionLookupTable_host, lookupTableNumCoefficients*lookupTableResolution*sizeof(complex<double>), hipMemcpyHostToDevice) != hipSuccess)
		{	cout << "\tMemcpy error\n";	exit(1);	}

	delete [] generatingFunctionLookupTable_host;
}

void ChebyshevSolver::destroyLookupTableGPU(){
	if(isTalkative)
		cout << "ChebyshevSolver::destroyLookupTableGPU\n";

	if(generatingFunctionLookupTable_device == NULL){
		cout << "Error: No lookup table loaded onto GPU.\n";
		exit(1);
	}

	hipFree(generatingFunctionLookupTable_device);
	generatingFunctionLookupTable_device = NULL;
}

void ChebyshevSolver::generateGreensFunctionGPU(complex<double> *greensFunction, complex<double> *coefficients){
	if(isTalkative)
		cout << "ChebyshevSolver::generateGreensFunctionGPU\n";

	if(generatingFunctionLookupTable_device == NULL){
		cout << "Error: No lookup table loaded onto GPU.\n";
		exit(1);
	}

	for(int e = 0; e < lookupTableResolution; e++)
		greensFunction[e] = 0.;

	complex<double> *greensFunction_device;
	complex<double> *coefficients_device;

	if(hipMalloc((void**)&greensFunction_device, lookupTableResolution*sizeof(complex<double>))  != hipSuccess)
		{	cout << "\tMalloc error\n";	exit(1);	}
	if(hipMalloc((void**)&coefficients_device, lookupTableNumCoefficients*sizeof(complex<double>))  != hipSuccess)
		{	cout << "\tMalloc error\n";	exit(1);	}

	if(hipMemcpy(greensFunction_device, greensFunction, lookupTableResolution*sizeof(complex<double>), hipMemcpyHostToDevice) != hipSuccess)
		{	cout << "\tMemcpy error\n";	exit(1);	}
	if(hipMemcpy(coefficients_device, coefficients, lookupTableNumCoefficients*sizeof(complex<double>), hipMemcpyHostToDevice) != hipSuccess)
		{	cout << "\tMemcpy error\n";	exit(1);	}

	int block_size = 1024;
	int num_blocks = lookupTableResolution/block_size + (lookupTableResolution%block_size == 0 ? 0:1);

	if(isTalkative){
		cout << "\tCUDA Block size: " << block_size << "\n";
		cout << "\tCUDA Num blocks: " << num_blocks << "\n";
	}

	calculateGreensFunction <<< num_blocks, block_size>>> ((hipDoubleComplex*)greensFunction_device,
								(hipDoubleComplex*)coefficients_device,
								(hipDoubleComplex*)generatingFunctionLookupTable_device,
								lookupTableNumCoefficients,
								lookupTableResolution);

	if(hipMemcpy(greensFunction, greensFunction_device, lookupTableResolution*sizeof(complex<double>), hipMemcpyDeviceToHost) != hipSuccess)
		{	cout << "\tMemcpy error\n";	exit(1);	}

	hipFree(greensFunction_device);
	hipFree(coefficients_device);
}

};	//End of namespace TBTK
